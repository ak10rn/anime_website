
#include <hip/hip_runtime.h>
#include<stdio.h>
// #include<cuda.h>

int n=100;

__global__ void VecAddKernel(float *d_A, float *d_B, float *d_C, int n){
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	if(i<n){
		d_C[i]=d_A[i]+d_B[i];
	}
}

void addWithCuda(float *A, float *B, float *C,int n){

	float *d_A, *d_B, *d_C;
	int size = n*sizeof(float);
	
	//Device Memory Allocation
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);
	
	//Transfer of data from host to device
	hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
	hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
	
	//Calling to Kernel
	VecAddKernel<<<ceil(100/16.0),16>>> (d_A,d_B,d_C,n);
	
	//Transfer of data from Device to Host
	hipMemcpy(C,d_C,size, hipMemcpyDeviceToHost);
}

int main(){
	float *A,*B,*C;
	int n;
	printf("Enter the size of Vector");
	
	scanf("%d",&n);
	A=(float*)malloc(n*sizeof(float));
	B=(float*)malloc(n*sizeof(float));
	C=(float*)malloc(n*sizeof(float));
	
	for(int i=0;i<n;i++){
		A[i]=i;
		B[i]=i*i;
	}
	
	addWithCuda(A,B,C,n);
	printf("The value of A+B i.e. C = \n{");
	
	for(int i=0;i<n;i++){
		printf("%f, ",C[i]);
	}
	
	printf("}\n");
	
	return 0;
}
